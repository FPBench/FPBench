#include "hip/hip_runtime.h"
#include <fenv.h>
#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>

__device__ double ex0() {
	double tmp;
	if (1.0 < 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex1() {
	double tmp;
	if (0.0 < 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex2() {
	double tmp;
	if (-1.0 < 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex3() {
	double tmp;
	if (0.0 < -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex4() {
	double tmp;
	if (1.0 < -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex5() {
	double tmp;
	if (-1.0 < 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex6() {
	double tmp;
	if (0.0 < 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex7() {
	double tmp;
	if (((double) HIP_NAN) < ((double) HIP_NAN)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex8() {
	double tmp;
	if (0.0 < 0.0 && 0.0 < 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex9() {
	double tmp;
	if (1.0 > 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex10() {
	double tmp;
	if (0.0 > 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex11() {
	double tmp;
	if (-1.0 > 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex12() {
	double tmp;
	if (0.0 > -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex13() {
	double tmp;
	if (1.0 > -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex14() {
	double tmp;
	if (-1.0 > 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex15() {
	double tmp;
	if (0.0 > 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex16() {
	double tmp;
	if (((double) HIP_NAN) > ((double) HIP_NAN)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex17() {
	double tmp;
	if (0.0 > 0.0 && 0.0 > 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex18() {
	double tmp;
	if (1.0 <= 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex19() {
	double tmp;
	if (0.0 <= 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex20() {
	double tmp;
	if (-1.0 <= 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex21() {
	double tmp;
	if (0.0 <= -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex22() {
	double tmp;
	if (1.0 <= -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex23() {
	double tmp;
	if (-1.0 <= 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex24() {
	double tmp;
	if (0.0 <= 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex25() {
	double tmp;
	if (((double) HIP_NAN) <= ((double) HIP_NAN)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex26() {
	double tmp;
	if (0.0 <= 0.0 && 0.0 <= 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex27() {
	double tmp;
	if (1.0 >= 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex28() {
	double tmp;
	if (0.0 >= 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex29() {
	double tmp;
	if (-1.0 >= 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex30() {
	double tmp;
	if (0.0 >= -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex31() {
	double tmp;
	if (1.0 >= -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex32() {
	double tmp;
	if (-1.0 >= 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex33() {
	double tmp;
	if (0.0 >= 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex34() {
	double tmp;
	if (((double) HIP_NAN) >= ((double) HIP_NAN)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex35() {
	double tmp;
	if (0.0 >= 0.0 && 0.0 >= 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex36() {
	double tmp;
	if (1.0 == 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex37() {
	double tmp;
	if (0.0 == 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex38() {
	double tmp;
	if (-1.0 == 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex39() {
	double tmp;
	if (0.0 == -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex40() {
	double tmp;
	if (1.0 == -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex41() {
	double tmp;
	if (-1.0 == 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex42() {
	double tmp;
	if (0.0 == 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex43() {
	double tmp;
	if (((double) HIP_NAN) == ((double) HIP_NAN)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex44() {
	double tmp;
	if (0.0 == 0.0 && 0.0 == 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex45() {
	double tmp;
	if (1.0 != 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex46() {
	double tmp;
	if (0.0 != 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex47() {
	double tmp;
	if (-1.0 != 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex48() {
	double tmp;
	if (0.0 != -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex49() {
	double tmp;
	if (1.0 != -1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex50() {
	double tmp;
	if (-1.0 != 1.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex51() {
	double tmp;
	if (0.0 != 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex52() {
	double tmp;
	if (((double) HIP_NAN) != ((double) HIP_NAN)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex53() {
	double tmp;
	if (0.0 == 0.0 && 0.0 == 0.0) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex54() {
	double tmp;
	if (TRUE && TRUE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex55() {
	double tmp;
	if (TRUE && FALSE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex56() {
	double tmp;
	if (FALSE && TRUE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex57() {
	double tmp;
	if (FALSE && FALSE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex58() {
	double tmp;
	if (TRUE && TRUE && TRUE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex59() {
	double tmp;
	if (TRUE && TRUE && FALSE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex60() {
	double tmp;
	if (TRUE || TRUE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex61() {
	double tmp;
	if (TRUE || FALSE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex62() {
	double tmp;
	if (FALSE || TRUE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex63() {
	double tmp;
	if (FALSE || FALSE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex64() {
	double tmp;
	if (FALSE || FALSE || FALSE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex65() {
	double tmp;
	if (FALSE || FALSE || TRUE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex66() {
	double tmp;
	if (!TRUE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex67() {
	double tmp;
	if (!FALSE) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex68() {
	double tmp;
	if (isinf(0.0)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex69() {
	double tmp;
	if (isinf(1.0)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex70() {
	double tmp;
	if (isinf(((double) HIP_INF))) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex71() {
	double tmp;
	if (isinf(((double) HIP_NAN))) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex72() {
	double tmp;
	if (isnan(0.0)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex73() {
	double tmp;
	if (isnan(1.0)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex74() {
	double tmp;
	if (isnan(((double) HIP_INF))) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex75() {
	double tmp;
	if (isnan(((double) HIP_NAN))) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex76() {
	double tmp;
	if (isfinite(0.0)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex77() {
	double tmp;
	if (isfinite(1.0)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex78() {
	double tmp;
	if (isfinite(((double) HIP_INF))) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex79() {
	double tmp;
	if (isfinite(((double) HIP_NAN))) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex80() {
	double tmp;
	if (isnormal(0.0)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex81() {
	double tmp;
	if (isnormal(1.0)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex82() {
	double tmp;
	if (isnormal(((double) HIP_INF))) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex83() {
	double tmp;
	if (isnormal(((double) HIP_NAN))) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex84() {
	double tmp;
	if (signbit(0.0)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex85() {
	double tmp;
	if (signbit(1.0)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

__device__ double ex86() {
	double tmp;
	if (signbit(-1.0)) {
		tmp = 1.0;
	} else {
		tmp = 0.0;
	}
	return tmp;
}

