#include <fenv.h>
#include <hip/hip_math_constants.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
__device__ double e = 2.71828182845904523536;

__device__ double ex0(double a, double b) {
	return (((333.75 * pow(b, 6.0)) + (pow(a, 2.0) * (((((11.0 * pow(a, 2.0)) * pow(b, 2.0)) - pow(b, 6.0)) - (121.0 * pow(b, 4.0))) - 2.0))) + (5.5 * pow(b, 8.0))) + (a / (2.0 * b));
}

__device__ double ex1(double a, double b) {
	double b2 = b * b;
	double b4 = b2 * b2;
	double b6 = b4 * b2;
	double b8 = b4 * b4;
	double a2 = a * a;
	double firstexpr = ((((11.0 * a2) * b2) - b6) - (121.0 * b4)) - 2.0;
	return (((333.75 * b6) + (a2 * firstexpr)) + (5.5 * b8)) + (a / (2.0 * b));
}

__device__ double ex2(double a, double b) {
	double b2 = b * b;
	double b4 = b2 * b2;
	double b6 = b4 * b2;
	double b8 = b4 * b4;
	double a2 = a * a;
	double firstexpr = (((11.0 * a2) * b2) - (121.0 * b4)) - 2.0;
	return ((((333.75 - a2) * b6) + (a2 * firstexpr)) + (5.5 * b8)) + (a / (2.0 * b));
}

